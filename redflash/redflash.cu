#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include <common.h>
#include "redflash.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 origin;
    float3 direction;

    float3 emission;
    float3 albedo;

    unsigned int seed;
    int depth;
    int done;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;


RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        float3 accumulation = make_float3(0.0f);
        float3 reflectance  = make_float3(1.0f);

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(int i = 0; i < 10; i++)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            accumulation += reflectance * prd.emission;
            reflectance *= prd.albedo * 1.0/*current_reflectance*/;

            if (prd.done)
            {
                // We have hit the background or a luminaire
                break;
            }

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;

            prd.depth++;
        }

        result += accumulation;
        seed = prd.seed;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.emission = emission_color;
    current_prd.albedo   = make_float3(0, 0, 0);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );


RT_PROGRAM void diffuse()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction + ffnormal * scene_epsilon * 100.0;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.albedo   = diffuse_color;
    current_prd.emission = make_float3(0.0);

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        ParallelogramLight light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L     = normalize(light_pos - hitpoint);
        const float  nDl   = dot( ffnormal, L );
        const float  LnDl  = dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = length(cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                current_prd.emission += light.emission * weight;
            }
        }
    }
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
    float theta = atan2f(ray.direction.x, ray.direction.z);
    float phi = M_PIf * 0.5f - acosf(ray.direction.y);
    float u = (theta + M_PIf) * (0.5f * M_1_PIf);
    float v = 0.5f * (1.0f + sin(phi));
    current_prd.emission = make_float3(tex2D(envmap, u, v));
    current_prd.albedo = make_float3(0.0);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Raymarching
//
//-----------------------------------------------------------------------------

#include <optix_world.h>

rtDeclareVariable(float3, center, , );
rtDeclareVariable(float3, size, , );
rtDeclareVariable(int, lgt_instance, , ) = {0};
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, );

float dMenger(float3 z0, float3 offset, float scale) {
    float4 z = make_float4(z0, 1.0);
    for (int n = 0; n < 4; n++) {
        // z = abs(z);
        z.x = abs(z.x);
        z.y = abs(z.y);
        z.z = abs(z.z);
        z.w = abs(z.w);

        // if (z.x < z.y) z.xy = z.yx;
        if (z.x < z.y)
        {
            float x = z.x;
            z.x = z.y;
            z.y = x;
        }

        // if (z.x < z.z) z.xz = z.zx;
        if (z.x < z.z)
        {
            float x = z.x;
            z.x = z.z;
            z.z = x;
        }

        // if (z.y < z.z) z.yz = z.zy;
        if (z.y < z.z)
        {
            float y = z.y;
            z.y = z.z;
            z.z = y;
        }

        z *= scale;
        // z.xyz -= offset * (scale - 1.0);
        z.x -= offset.x * (scale - 1.0);
        z.y -= offset.y * (scale - 1.0);
        z.z -= offset.z * (scale - 1.0);

        if (z.z < -0.5 * offset.z * (scale - 1.0))
            z.z += offset.z * (scale - 1.0);
    }
    // return (length(max(abs(z.xyz) - make_float3(1.0, 1.0, 1.0), 0.0)) - 0.05) / z.w;
    return (length(make_float3(max(abs(z.x) - 1.0, 0.0), max(abs(z.y) - 1.0, 0.0), max(abs(z.z) - 1.0, 0.0))) - 0.05) / z.w;
}

float3 get_xyz(float4 p)
{
    return make_float3(p.x, p.y, p.z);
}

// not work...
void set_xyz(float4 &a, float3 b)
{
    a.x = b.x;
    a.y = b.y;
    a.x = b.z;
}

float dMandelFast(float3 p, float scale, int n) {
    float4 q0 = make_float4(p, 1.);
    float4 q = q0;

    for (int i = 0; i < n; i++) {
        // q.xyz = clamp(q.xyz, -1.0, 1.0) * 2.0 - q.xyz;
        // set_xyz(q, clamp(get_xyz(q), -1.0, 1.0) * 2.0 - get_xyz(q));
        float4 tmp = clamp(q, -1.0, 1.0) * 2.0 - q;
        q.x = tmp.x;
        q.y = tmp.y;
        q.z = tmp.z;

        // q = q * scale / clamp( dot( q.xyz, q.xyz ), 0.3, 1.0 ) + q0;
        float3 q_xyz = get_xyz(q);
        q = q * scale / clamp(dot(q_xyz, q_xyz), 0.3, 1.0) + q0;
    }

    // return length( q.xyz ) / abs( q.w );
    return length(get_xyz(q)) / abs(q.w);
}

float map(float3 p)
{
    //return length(p - center) - 100.0;

    float scale = 100 * 0.3;
    // f((p - position) / scale) * scale;
    // return dMenger((p - center) / scale, make_float3(1.23, 1.65, 1.45), 2.56) * scale;
    // return dMenger((p - center) / scale, make_float3(1, 1, 1), 3.1) * scale;
    return dMandelFast((p - center) / scale, 2.76, 20) * scale;
}

#define calcNormal(p, dFunc, eps) normalize(\
    make_float3( eps, -eps, -eps) * dFunc(p + make_float3( eps, -eps, -eps)) + \
    make_float3(-eps, -eps,  eps) * dFunc(p + make_float3(-eps, -eps,  eps)) + \
    make_float3(-eps,  eps, -eps) * dFunc(p + make_float3(-eps,  eps, -eps)) + \
    make_float3( eps,  eps,  eps) * dFunc(p + make_float3( eps,  eps,  eps)))

float3 calcNormalBasic(float3 p, float eps)
{
    return normalize(make_float3(
        map(p + make_float3(eps, 0.0, 0.0)) - map(p + make_float3(-eps, 0.0, 0.0)),
        map(p + make_float3(0.0, eps, 0.0)) - map(p + make_float3(0.0, -eps, 0.0)),
        map(p + make_float3(0.0, 0.0, eps)) - map(p + make_float3(0.0, 0.0, -eps))
    ));
}

RT_PROGRAM void intersect(int primIdx)
{
    const float EPS = scene_epsilon;
    float t = ray.tmin, d = 0.0;
    float3 p = ray.origin;

    for (int i = 0; i < 300; i++)
    {
        p = ray.origin + t * ray.direction;
        d = map(p);
        t += d;
        if (abs(d) < EPS || t > ray.tmax)
        {
            break;
        }
    }

    if (t < ray.tmax && rtPotentialIntersection(t))
    {
        shading_normal = geometric_normal = calcNormal(p, map, scene_epsilon);
        texcoord = make_float3(p.x, p.y, 0);
        lgt_idx = lgt_instance;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bounds(int, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = center - size;
    aabb->m_max = center + size;
}