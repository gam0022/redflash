#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include "redflash.h"
#include "random.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float3, center, , );
rtDeclareVariable(float3, local_scale, , );
rtDeclareVariable(float3, aabb_min, , );
rtDeclareVariable(float3, aabb_max, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

float dMenger(float3 z0, float3 offset, float scale) {
    float4 z = make_float4(z0, 1.0);
    for (int n = 0; n < 4; n++) {
        // z = abs(z);
        z.x = abs(z.x);
        z.y = abs(z.y);
        z.z = abs(z.z);
        z.w = abs(z.w);

        // if (z.x < z.y) z.xy = z.yx;
        if (z.x < z.y)
        {
            float x = z.x;
            z.x = z.y;
            z.y = x;
        }

        // if (z.x < z.z) z.xz = z.zx;
        if (z.x < z.z)
        {
            float x = z.x;
            z.x = z.z;
            z.z = x;
        }

        // if (z.y < z.z) z.yz = z.zy;
        if (z.y < z.z)
        {
            float y = z.y;
            z.y = z.z;
            z.z = y;
        }

        z *= scale;
        // z.xyz -= offset * (scale - 1.0);
        z.x -= offset.x * (scale - 1.0);
        z.y -= offset.y * (scale - 1.0);
        z.z -= offset.z * (scale - 1.0);

        if (z.z < -0.5 * offset.z * (scale - 1.0))
            z.z += offset.z * (scale - 1.0);
    }
    // return (length(max(abs(z.xyz) - make_float3(1.0, 1.0, 1.0), 0.0)) - 0.05) / z.w;
    return (length(make_float3(max(abs(z.x) - 1.0, 0.0), max(abs(z.y) - 1.0, 0.0), max(abs(z.z) - 1.0, 0.0))) - 0.05) / z.w;
}

float3 get_xyz(float4 p)
{
    return make_float3(p.x, p.y, p.z);
}

// not work...
void set_xyz(float4 &a, float3 b)
{
    a.x = b.x;
    a.y = b.y;
    a.x = b.z;
}

float dMandelFast(float3 p, float scale, int n) {
    float4 q0 = make_float4(p, 1.);
    float4 q = q0;

    for (int i = 0; i < n; i++) {
        // q.xyz = clamp(q.xyz, -1.0, 1.0) * 2.0 - q.xyz;
        // set_xyz(q, clamp(get_xyz(q), -1.0, 1.0) * 2.0 - get_xyz(q));
        float4 tmp = clamp(q, -1.0, 1.0) * 2.0 - q;
        q.x = tmp.x;
        q.y = tmp.y;
        q.z = tmp.z;

        // q = q * scale / clamp( dot( q.xyz, q.xyz ), 0.3, 1.0 ) + q0;
        float3 q_xyz = get_xyz(q);
        q = q * scale / clamp(dot(q_xyz, q_xyz), 0.3, 1.0) + q0;
    }

    // return length( q.xyz ) / abs( q.w );
    return length(get_xyz(q)) / abs(q.w);
}

float map(float3 p)
{
    // return dMenger((p - center) / local_scale, make_float3(1.23, 1.65, 1.45), 2.56) * local_scale;
    // return dMenger((p - center) / local_scale, make_float3(1, 1, 1), 3.1) * local_scale;
    return dMandelFast((p - center) / local_scale, 2.76, 20) * min(min(local_scale.x, local_scale.y), local_scale.z);
}

#define calcNormal(p, dFunc, eps) normalize(\
    make_float3( eps, -eps, -eps) * dFunc(p + make_float3( eps, -eps, -eps)) + \
    make_float3(-eps, -eps,  eps) * dFunc(p + make_float3(-eps, -eps,  eps)) + \
    make_float3(-eps,  eps, -eps) * dFunc(p + make_float3(-eps,  eps, -eps)) + \
    make_float3( eps,  eps,  eps) * dFunc(p + make_float3( eps,  eps,  eps)))

float3 calcNormalBasic(float3 p, float eps)
{
    return normalize(make_float3(
        map(p + make_float3(eps, 0.0, 0.0)) - map(p + make_float3(-eps, 0.0, 0.0)),
        map(p + make_float3(0.0, eps, 0.0)) - map(p + make_float3(0.0, -eps, 0.0)),
        map(p + make_float3(0.0, 0.0, eps)) - map(p + make_float3(0.0, 0.0, -eps))
    ));
}

RT_PROGRAM void intersect(int primIdx)
{
    float t = ray.tmin, d = 0.0;
    float3 p;

    // int loop_max = ray.tmin > scene_epsilon * 10.0f ? 10 : 300;

    for (int i = 0; i < 300; i++)
    {
        p = ray.origin + t * ray.direction;
        d = map(p);
        t += d;
        float eps = scene_epsilon * t;
        if (abs(d) < eps || t > ray.tmax)
        {
            break;
        }
    }

    if (t < ray.tmax && rtPotentialIntersection(t))
    {
        shading_normal = geometric_normal = calcNormal(p, map, scene_epsilon);
        texcoord = make_float3(p.x, p.y, 0);
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bounds(int, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = aabb_min;
    aabb->m_max = aabb_max;
}